#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include "legate_library.h"
#include "legateboost.h"
#include "../../cpp_utils/cpp_utils.h"
#include "../../cpp_utils/cpp_utils.cuh"
#include "core/comm/coll.h"
#include "build_tree.h"
#include <numeric>

#include <cuda/std/tuple>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <thrust/unique.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

namespace legateboost {

struct NodeBatch {
  int32_t node_idx_begin;
  int32_t node_idx_end;
  cuda::std::tuple<int32_t, int32_t>* instances_begin;
  cuda::std::tuple<int32_t, int32_t>* instances_end;
  __host__ __device__ std::size_t InstancesInBatch() const
  {
    return instances_end - instances_begin;
  }
  __host__ __device__ std::size_t NodesInBatch() const { return node_idx_end - node_idx_begin; }
};

class GradientQuantiser {
  IntegerGPair scale;

 public:
  struct GetAbsGPair {
    int num_outputs;
    legate::AccessorRO<double, 3> g;
    legate::AccessorRO<double, 3> h;
    __device__ GPair operator()(int n) const
    {
      legate::Point<3> p = {n / num_outputs, 0, n % num_outputs};
      return GPair{abs(g[p]), abs(h[p])};
    }
  };

  // Calculate scale from upper bound on data
  GradientQuantiser(legate::TaskContext context,
                    legate::AccessorRO<double, 3> g,
                    legate::AccessorRO<double, 3> h,
                    legate::Rect<3> g_shape,
                    hipStream_t stream)
  {
    auto thrust_alloc = ThrustAllocator(legate::Memory::GPU_FB_MEM);
    auto policy       = DEFAULT_POLICY(thrust_alloc).on(stream);
    auto counting     = thrust::make_counting_iterator(0);
    int num_outputs   = g_shape.hi[2] - g_shape.lo[2] + 1;
    std::size_t n     = (g_shape.hi[0] - g_shape.lo[0] + 1) * num_outputs;
    auto zip_gpair    = thrust::make_transform_iterator(counting, GetAbsGPair{num_outputs, g, h});
    GPair abs_sum =
      thrust::reduce(policy, zip_gpair, zip_gpair + n, GPair{0.0, 0.0}, thrust::plus<GPair>());
    SumAllReduce(context, reinterpret_cast<double*>(&abs_sum), 2);

    // We will quantise values between -max_int and max_int
    // Double precision can exactly represent integers in this range
    // So we can go back and forth between double and int64_t without overflow
    int64_t double_max_int = 1ll << 51;
    int64_t max_int =
      std::min<int64_t>(double_max_int, std::numeric_limits<IntegerGPair::value_type>::max());
    scale.grad = abs_sum.grad == 0 ? 1 : max_int / abs_sum.grad;
    scale.hess = abs_sum.hess == 0 ? 1 : max_int / abs_sum.hess;
  }

  __device__ IntegerGPair Quantise(GPair value) const
  {
    IntegerGPair result;
    result.grad = value.grad * scale.grad;
    result.hess = value.hess * scale.hess;
    return result;
  }

  __device__ GPair Dequantise(IntegerGPair value) const
  {
    GPair result;
    result.grad = double(value.grad) / scale.grad;
    result.hess = double(value.hess) / scale.hess;
    return result;
  }
};

__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  reduce_base_sums(legate::AccessorRO<double, 3> g,
                   legate::AccessorRO<double, 3> h,
                   size_t n_local_samples,
                   int64_t sample_offset,
                   legate::Buffer<IntegerGPair, 2> node_sums,
                   size_t n_outputs,
                   GradientQuantiser quantiser)
{
  typedef hipcub::BlockReduce<IntegerGPair, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int32_t output = blockIdx.y;

  int64_t sample_id = threadIdx.x + blockDim.x * blockIdx.x;

  double grad = sample_id < n_local_samples ? g[{sample_id + sample_offset, 0, output}] : 0.0;
  double hess = sample_id < n_local_samples ? h[{sample_id + sample_offset, 0, output}] : 0.0;

  IntegerGPair blocksum = BlockReduce(temp_storage).Sum(quantiser.Quantise({grad, hess}));

  if (threadIdx.x == 0) {
    atomicAdd(
      reinterpret_cast<Histogram<IntegerGPair>::atomic_add_type*>(&node_sums[{0, output}].grad),
      blocksum.grad);
    atomicAdd(
      reinterpret_cast<Histogram<IntegerGPair>::atomic_add_type*>(&node_sums[{0, output}].hess),
      blocksum.hess);
  }
}

template <typename TYPE, int ELEMENTS_PER_THREAD, int FEATURES_PER_BLOCK>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  fill_histogram(legate::AccessorRO<TYPE, 3> X,
                 size_t n_features,
                 int64_t sample_offset,
                 legate::AccessorRO<double, 3> g,
                 legate::AccessorRO<double, 3> h,
                 size_t n_outputs,
                 SparseSplitProposals<TYPE> split_proposals,
                 NodeBatch batch,
                 Histogram<IntegerGPair> histogram,
                 legate::Buffer<IntegerGPair, 2> node_sums,
                 GradientQuantiser quantiser)
{
  // block dimensions are (THREADS_PER_BLOCK, 1, 1)
  // each thread processes ELEMENTS_PER_THREAD samples and FEATURES_PER_BLOCK features
  // the features to process are defined via blockIdx.y

  // further improvements:
  // * quantize values to work with int instead of double

#pragma unroll
  for (int32_t elementIdx = 0; elementIdx < ELEMENTS_PER_THREAD; ++elementIdx) {
    // within each iteration a (THREADS_PER_BLOCK, FEATURES_PER_BLOCK)-block of
    // data from X is processed.

    // check if thread has actual work to do
    int64_t idx      = (blockIdx.x + elementIdx * gridDim.x) * THREADS_PER_BLOCK + threadIdx.x;
    bool validThread = idx < batch.InstancesInBatch();
    if (!validThread) continue;
    auto [sampleNode, localSampleId] = batch.instances_begin[idx];
    int64_t globalSampleId           = localSampleId + sample_offset;

    bool computeHistogram = ComputeHistogramBin(
      sampleNode, node_sums, histogram.ContainsNode(BinaryTree::Parent(sampleNode)));

    for (int32_t output = 0; output < n_outputs; output++) {
      auto gpair_quantised =
        quantiser.Quantise({g[{globalSampleId, 0, output}], h[{globalSampleId, 0, output}]});
      for (int32_t featureIdx = 0; featureIdx < FEATURES_PER_BLOCK; featureIdx++) {
        int32_t feature = featureIdx + blockIdx.y * FEATURES_PER_BLOCK;
        if (computeHistogram && feature < n_features) {
          auto x_value = X[{globalSampleId, feature, 0}];
          auto bin_idx = split_proposals.FindBin(x_value, feature);

          // bin_idx is the first sample that is larger than x_value
          if (bin_idx != SparseSplitProposals<TYPE>::NOT_FOUND) {
            Histogram<IntegerGPair>::atomic_add_type* addPosition =
              reinterpret_cast<Histogram<IntegerGPair>::atomic_add_type*>(
                &histogram[{sampleNode, output, bin_idx}]);
            atomicAdd(addPosition, gpair_quantised.grad);
            atomicAdd(addPosition + 1, gpair_quantised.hess);
          }
        }
      }
    }
  }
}

template <typename T>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK)
  scan_kernel(Histogram<IntegerGPair> histogram,
              legate::Buffer<IntegerGPair, 2> node_sums,
              int n_features,
              int n_outputs,
              const SparseSplitProposals<T> split_proposals,
              NodeBatch batch)

{
  auto warp      = cg::tiled_partition<32>(cg::this_thread_block());
  int rank       = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  auto num_nodes = batch.NodesInBatch();
  int i          = rank / num_nodes;
  int j          = rank % num_nodes;

  // Specialize WarpScan for type int
  typedef hipcub::WarpScan<IntegerGPair> WarpScan;

  __shared__ typename WarpScan::TempStorage temp_storage[THREADS_PER_BLOCK / 32];

  int scan_node_idx = batch.node_idx_begin + j;
  int parent        = BinaryTree::Parent(scan_node_idx);
  // Exit if we didn't compute this histogram
  if (!ComputeHistogramBin(scan_node_idx, node_sums, histogram.ContainsNode(parent))) return;
  if (i >= n_features || scan_node_idx >= batch.node_idx_end) return;

  int feature_idx                   = i;
  auto [feature_begin, feature_end] = split_proposals.FeatureRange(feature_idx);
  int num_bins                      = feature_end - feature_begin;
  int num_tiles                     = (num_bins + warp.num_threads() - 1) / warp.num_threads();

  for (int output = 0; output < n_outputs; output++) {
    IntegerGPair aggregate;
    // Scan left side
    for (int tile_idx = 0; tile_idx < num_tiles; tile_idx++) {
      int bin_idx              = feature_begin + tile_idx * warp.num_threads() + warp.thread_rank();
      bool thread_participates = bin_idx < feature_end;
      auto e =
        thread_participates ? histogram[{scan_node_idx, output, bin_idx}] : IntegerGPair{0, 0};
      IntegerGPair tile_aggregate;
      WarpScan(temp_storage[threadIdx.x / warp.num_threads()]).InclusiveSum(e, e, tile_aggregate);
      __syncwarp();
      if (thread_participates) { histogram[{scan_node_idx, output, bin_idx}] = e + aggregate; }
      aggregate += tile_aggregate;
    }
  }

  // This node has no sibling we are finished
  if (scan_node_idx == 0) return;

  int sibling_node_idx = BinaryTree::Sibling(scan_node_idx);

  // The sibling did not compute a histogram
  // Do the subtraction trick using the histogram we just computed in the previous step
  if (!ComputeHistogramBin(sibling_node_idx, node_sums, histogram.ContainsNode(parent))) {
    for (int output = 0; output < n_outputs; output++) {
      // Infer right side
      for (int bin_idx = feature_begin + warp.thread_rank(); bin_idx < feature_end;
           bin_idx += warp.num_threads()) {
        auto scanned_sum = histogram[{scan_node_idx, output, bin_idx}];
        auto parent_sum  = histogram[{BinaryTree::Parent(scan_node_idx), output, bin_idx}];
        auto other_sum   = parent_sum - scanned_sum;
        histogram[{sibling_node_idx, output, bin_idx}] = other_sum;
      }
    }
  }
}

// Key/value pair to simplify reduction
struct GainFeaturePair {
  double gain;
  int feature;
  int feature_sample_idx;

  __device__ void operator=(const GainFeaturePair& other)
  {
    gain               = other.gain;
    feature            = other.feature;
    feature_sample_idx = other.feature_sample_idx;
  }

  __device__ bool operator==(const GainFeaturePair& other) const
  {
    return gain == other.gain && feature == other.feature &&
           feature_sample_idx == other.feature_sample_idx;
  }

  __device__ bool operator>(const GainFeaturePair& other) const { return gain > other.gain; }

  __device__ bool operator<(const GainFeaturePair& other) const { return gain < other.gain; }
};

template <typename TYPE>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  perform_best_split(Histogram<IntegerGPair> histogram,
                     size_t n_features,
                     size_t n_outputs,
                     SparseSplitProposals<TYPE> split_proposals,
                     double eps,
                     double alpha,
                     legate::Buffer<double, 2> tree_leaf_value,
                     legate::Buffer<IntegerGPair, 2> node_sums,
                     legate::Buffer<int32_t, 1> tree_feature,
                     legate::Buffer<double, 1> tree_split_value,
                     legate::Buffer<double, 1> tree_gain,
                     NodeBatch batch,
                     GradientQuantiser quantiser)
{
  // using one block per (level) node to have blockwise reductions
  int node_id = batch.node_idx_begin + blockIdx.x;

  typedef hipcub::BlockReduce<GainFeaturePair, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  __shared__ double node_best_gain;
  __shared__ int node_best_feature;
  __shared__ int node_best_bin_idx;

  double thread_best_gain = 0;
  int thread_best_feature = -1;
  int thread_best_bin_idx = -1;

  for (int feature_id = 0; feature_id < n_features; feature_id++) {
    auto [feature_start, feature_end] = split_proposals.FeatureRange(feature_id);

    for (int bin_idx = feature_start + threadIdx.x; bin_idx < feature_end; bin_idx += blockDim.x) {
      double gain = 0;
      for (int output = 0; output < n_outputs; ++output) {
        auto node_sum   = node_sums[{node_id, output}];
        auto left_sum   = histogram[{node_id, output, bin_idx}];
        auto right_sum  = node_sum - left_sum;
        auto [G, H]     = quantiser.Dequantise(node_sum);
        auto [G_L, H_L] = quantiser.Dequantise(left_sum);
        auto [G_R, H_R] = quantiser.Dequantise(right_sum);

        if (H_L <= 0.0 || H_R <= 0.0) {
          gain = 0;
          break;
        }
        double reg = std::max(eps, alpha);  // Regularisation term
        gain += 0.5 * ((G_L * G_L) / (H_L + reg) + (G_R * G_R) / (H_R + reg) - (G * G) / (H + reg));
      }
      if (gain > thread_best_gain) {
        thread_best_gain    = gain;
        thread_best_feature = feature_id;
        thread_best_bin_idx = bin_idx;
      }
    }
  }

  // SYNC BEST GAIN TO FULL BLOCK/NODE
  GainFeaturePair thread_best_pair{thread_best_gain, thread_best_feature, thread_best_bin_idx};
  GainFeaturePair node_best_pair =
    BlockReduce(temp_storage).Reduce(thread_best_pair, hipcub::Max(), THREADS_PER_BLOCK);
  if (threadIdx.x == 0) {
    node_best_gain    = node_best_pair.gain;
    node_best_feature = node_best_pair.feature;
    node_best_bin_idx = node_best_pair.feature_sample_idx;
  }
  __syncthreads();

  if (node_best_gain > eps) {
    for (int output = threadIdx.x; output < n_outputs; output += blockDim.x) {
      auto node_sum   = node_sums[{node_id, output}];
      auto left_sum   = histogram[{node_id, output, node_best_bin_idx}];
      auto right_sum  = node_sum - left_sum;
      auto [G_L, H_L] = quantiser.Dequantise(left_sum);
      auto [G_R, H_R] = quantiser.Dequantise(right_sum);

      int left_child                         = BinaryTree::LeftChild(node_id);
      int right_child                        = BinaryTree::RightChild(node_id);
      tree_leaf_value[{left_child, output}]  = CalculateLeafValue(G_L, H_L, alpha);
      tree_leaf_value[{right_child, output}] = CalculateLeafValue(G_R, H_R, alpha);
      node_sums[{left_child, output}]        = left_sum;
      node_sums[{right_child, output}]       = right_sum;

      if (output == 0) {
        tree_feature[node_id]     = node_best_feature;
        tree_split_value[node_id] = split_proposals.split_proposals[node_best_bin_idx];
        tree_gain[node_id]        = node_best_gain;
      }
    }
  }
}

namespace {

struct Tree {
  template <typename THRUST_POLICY>
  Tree(int max_nodes, int num_outputs, hipStream_t stream, const THRUST_POLICY& thrust_exec_policy)
    : num_outputs(num_outputs), max_nodes(max_nodes), stream(stream)
  {
    leaf_value  = legate::create_buffer<double, 2>({max_nodes, num_outputs});
    feature     = legate::create_buffer<int32_t, 1>(max_nodes);
    split_value = legate::create_buffer<double, 1>(max_nodes);
    gain        = legate::create_buffer<double, 1>(max_nodes);
    node_sums   = legate::create_buffer<IntegerGPair, 2>({max_nodes, num_outputs});
    thrust::fill(thrust_exec_policy,
                 leaf_value.ptr({0, 0}),
                 leaf_value.ptr({0, 0}) + max_nodes * num_outputs,
                 0.0);
    thrust::fill(thrust_exec_policy, feature.ptr(0), feature.ptr(0) + max_nodes, -1);
    thrust::fill(thrust_exec_policy, split_value.ptr(0), split_value.ptr(0) + max_nodes, 0.0);
    thrust::fill(thrust_exec_policy, gain.ptr(0), gain.ptr(0) + max_nodes, 0.0);
    thrust::fill(thrust_exec_policy,
                 node_sums.ptr({0, 0}),
                 node_sums.ptr({0, 0}) + max_nodes * num_outputs,
                 IntegerGPair{0, 0});
  }

  template <typename T, int DIM, typename ThrustPolicyT>
  void WriteOutput(legate::PhysicalStore out,
                   const legate::Buffer<T, DIM> x,
                   const ThrustPolicyT& policy)
  {
    // Write a tile of x to the output
    const legate::Rect<DIM> out_shape = out.shape<DIM>();
    auto out_acc                      = out.write_accessor<T, DIM>();
    thrust::for_each_n(policy,
                       UnravelIter(out_shape),
                       out_shape.volume(),
                       [=] __device__(const legate::Point<DIM>& p) { out_acc[p] = x[p]; });
  }

  template <typename ThrustPolicyT>
  void WriteTreeOutput(legate::TaskContext context,
                       const ThrustPolicyT& policy,
                       GradientQuantiser quantiser)
  {
    WriteOutput(context.output(0).data(), leaf_value, policy);
    WriteOutput(context.output(1).data(), feature, policy);
    WriteOutput(context.output(2).data(), split_value, policy);
    WriteOutput(context.output(3).data(), gain, policy);

    // Dequantise and write the hessians
    auto hessian                        = context.output(4).data();
    const legate::Rect<2> hessian_shape = hessian.shape<2>();
    auto hessian_acc                    = hessian.write_accessor<double, 2>();
    auto node_sums                      = this->node_sums;  // Dont let device lambda capture this
    thrust::for_each_n(
      policy, UnravelIter(hessian_shape), hessian_shape.volume(), [=] __device__(auto p) {
        hessian_acc[p] = quantiser.Dequantise(node_sums[p]).hess;
      });
    CHECK_CUDA_STREAM(stream);
  }

  legate::Buffer<double, 2> leaf_value;
  legate::Buffer<int32_t, 1> feature;
  legate::Buffer<double, 1> split_value;
  legate::Buffer<double, 1> gain;
  legate::Buffer<IntegerGPair, 2> node_sums;
  const int num_outputs;
  const int max_nodes;
  hipStream_t stream;
};

// Randomly sample split_samples rows from X
// Use nccl to share the samples with all workers
// Remove any duplicates
// Return sparse matrix of split samples for each feature
template <typename T>
SparseSplitProposals<T> SelectSplitSamples(legate::TaskContext context,
                                           legate::AccessorRO<T, 3> X,
                                           legate::Rect<3> X_shape,
                                           int split_samples,
                                           int seed,
                                           int64_t dataset_rows,
                                           hipStream_t stream)
{
  auto thrust_alloc = ThrustAllocator(legate::Memory::GPU_FB_MEM);
  auto policy       = DEFAULT_POLICY(thrust_alloc).on(stream);
  int num_features  = X_shape.hi[1] - X_shape.lo[1] + 1;
  // Randomly choose split_samples rows
  auto row_samples = legate::create_buffer<int64_t, 1>(split_samples);
  auto counting    = thrust::make_counting_iterator(0);
  thrust::transform(
    policy, counting, counting + split_samples, row_samples.ptr(0), [=] __device__(int64_t idx) {
      thrust::default_random_engine eng(seed);
      thrust::uniform_int_distribution<int64_t> dist(0, dataset_rows - 1);
      eng.discard(idx);
      return dist(eng);
    });
  auto draft_proposals = legate::create_buffer<T, 2>({num_features, split_samples});

  // fill with local data
  LaunchN(num_features * split_samples, stream, [=] __device__(auto idx) {
    auto i                  = idx / num_features;
    auto j                  = idx % num_features;
    auto row                = row_samples[i];
    bool has_data           = row >= X_shape.lo[0] && row <= X_shape.hi[0];
    draft_proposals[{j, i}] = has_data ? X[{row, j, 0}] : T(0);
  });

  // Sum reduce over all workers
  SumAllReduce(context, draft_proposals.ptr({0, 0}), num_features * split_samples, stream);

  CHECK_CUDA_STREAM(stream);

  // Condense split samples to unique values
  // First sort the samples
  auto keys = legate::create_buffer<int32_t, 1>(num_features * split_samples);
  thrust::transform(
    policy, counting, counting + num_features * split_samples, keys.ptr(0), [=] __device__(int i) {
      return i / split_samples;
    });

  // Segmented sort
  auto begin =
    thrust::make_zip_iterator(thrust::make_tuple(keys.ptr(0), draft_proposals.ptr({0, 0})));
  thrust::sort(policy, begin, begin + num_features * split_samples, [] __device__(auto a, auto b) {
    if (thrust::get<0>(a) != thrust::get<0>(b)) { return thrust::get<0>(a) < thrust::get<0>(b); }
    return thrust::get<1>(a) < thrust::get<1>(b);
  });

  // Extract the unique values
  auto out_keys        = legate::create_buffer<int32_t, 1>(num_features * split_samples);
  auto split_proposals = legate::create_buffer<T, 1>(num_features * split_samples);
  auto key_val =
    thrust::make_zip_iterator(thrust::make_tuple(keys.ptr(0), draft_proposals.ptr({0, 0})));
  auto out_iter =
    thrust::make_zip_iterator(thrust::make_tuple(out_keys.ptr(0), split_proposals.ptr(0)));
  auto result =
    thrust::unique_copy(policy, key_val, key_val + num_features * split_samples, out_iter);
  auto n_unique = thrust::distance(out_iter, result);
  // Count the unique values for each feature
  auto row_pointers = legate::create_buffer<int32_t, 1>(num_features + 1);
  CHECK_CUDA(hipMemsetAsync(row_pointers.ptr(0), 0, (num_features + 1) * sizeof(int32_t), stream));

  thrust::reduce_by_key(policy,
                        out_keys.ptr(0),
                        out_keys.ptr(0) + n_unique,
                        thrust::make_constant_iterator(1),
                        thrust::make_discard_iterator(),
                        row_pointers.ptr(1));
  // Scan the counts to get the row pointers for a CSR matrix
  thrust::inclusive_scan(
    policy, row_pointers.ptr(1), row_pointers.ptr(1) + num_features, row_pointers.ptr(1));

  CHECK_CUDA(hipStreamSynchronize(stream));
  row_samples.destroy();
  draft_proposals.destroy();
  out_keys.destroy();
  return SparseSplitProposals<T>(split_proposals, row_pointers, num_features, n_unique);
}

// Can't put a device lambda in constructor so make this a function
void FillPositions(legate::Buffer<cuda::std::tuple<int32_t, int32_t>> sorted_positions,
                   std::size_t num_rows,
                   hipStream_t stream)
{
  LaunchN(num_rows, stream, [=] __device__(std::size_t idx) {
    sorted_positions[idx] = cuda::std::make_tuple(0, idx);
  });
}

template <typename T>
struct TreeBuilder {
  TreeBuilder(int32_t num_rows,
              int32_t num_features,
              int32_t num_outputs,
              hipStream_t stream,
              int32_t max_nodes,
              int32_t max_depth,
              SparseSplitProposals<T> split_proposals,
              GradientQuantiser quantiser)
    : num_rows(num_rows),
      num_features(num_features),
      num_outputs(num_outputs),
      stream(stream),
      max_nodes(max_nodes),
      split_proposals(split_proposals),
      quantiser(quantiser)
  {
    sorted_positions = legate::create_buffer<cuda::std::tuple<int32_t, int32_t>>(num_rows);
    FillPositions(sorted_positions, num_rows, stream);

    // Calculate the number of node histograms we are willing to cache
    // User a fixed reasonable upper bound on memory usage
    // CAUTION: all workers MUST have the same max_batch_size
    // Therefore we don't try to calculate this based on available memory
    const std::size_t max_bytes      = std::pow(10, 9);  // 1 GB
    const std::size_t bytes_per_node = num_outputs * split_proposals.histogram_size * sizeof(GPair);
    const std::size_t max_histogram_nodes = std::max(1ul, max_bytes / bytes_per_node);
    int depth                             = 0;
    while (BinaryTree::LevelEnd(depth + 1) <= max_histogram_nodes && depth <= max_depth) depth++;
    histogram      = Histogram<IntegerGPair>(BinaryTree::LevelBegin(0),
                                        BinaryTree::LevelEnd(depth),
                                        num_outputs,
                                        split_proposals.histogram_size,
                                        stream);
    max_batch_size = max_histogram_nodes;
  }

  template <typename TYPE>
  void UpdatePositions(Tree& tree, legate::AccessorRO<TYPE, 3> X, legate::Rect<3> X_shape)
  {
    auto tree_split_value_ptr = tree.split_value.ptr(0);
    auto tree_feature_ptr     = tree.feature.ptr(0);
    auto max_nodes_           = this->max_nodes;

    LaunchN(
      num_rows, stream, [=, sorted_positions = this->sorted_positions] __device__(size_t idx) {
        auto [pos, row] = sorted_positions[idx];

        if (pos < 0 || pos >= max_nodes_ || tree_feature_ptr[pos] == -1) {
          sorted_positions[idx] = cuda::std::make_tuple(-1, row);
          return;
        }
        double x_value        = X[{X_shape.lo[0] + (int64_t)row, tree_feature_ptr[pos], 0}];
        bool left             = x_value <= tree_split_value_ptr[pos];
        pos                   = left ? BinaryTree::LeftChild(pos) : BinaryTree::RightChild(pos);
        sorted_positions[idx] = cuda::std::make_tuple(pos, row);
      });
    CHECK_CUDA_STREAM(stream);
  }

  template <typename TYPE>
  void ComputeHistogram(Histogram<IntegerGPair> histogram,
                        legate::TaskContext context,
                        Tree& tree,
                        legate::AccessorRO<TYPE, 3> X,
                        legate::Rect<3> X_shape,
                        legate::AccessorRO<double, 3> g,
                        legate::AccessorRO<double, 3> h,
                        NodeBatch batch)
  {
    // TODO adjust kernel parameters dynamically
    constexpr size_t elements_per_thread = 8;
    constexpr size_t features_per_block  = 16;

    const size_t blocks_x =
      (batch.InstancesInBatch() + THREADS_PER_BLOCK * elements_per_thread - 1) /
      (THREADS_PER_BLOCK * elements_per_thread);
    const size_t blocks_y = (num_features + features_per_block - 1) / features_per_block;
    dim3 grid_shape       = dim3(blocks_x, blocks_y, 1);
    fill_histogram<TYPE, elements_per_thread, features_per_block>
      <<<grid_shape, THREADS_PER_BLOCK, 0, stream>>>(X,
                                                     num_features,
                                                     X_shape.lo[0],
                                                     g,
                                                     h,
                                                     num_outputs,
                                                     split_proposals,
                                                     batch,
                                                     histogram,
                                                     tree.node_sums,
                                                     quantiser);

    CHECK_CUDA_STREAM(stream);
    static_assert(sizeof(GPair) == 2 * sizeof(double), "GPair must be 2 doubles");
    SumAllReduce(context,
                 reinterpret_cast<Histogram<IntegerGPair>::value_type::value_type*>(
                   histogram.Ptr(batch.node_idx_begin)),
                 batch.NodesInBatch() * num_outputs * split_proposals.histogram_size * 2,
                 stream);

    const size_t warps_needed    = num_features * batch.NodesInBatch();
    const size_t warps_per_block = THREADS_PER_BLOCK / 32;
    const size_t blocks_needed   = (warps_needed + warps_per_block - 1) / warps_per_block;

    // Scan the histograms
    scan_kernel<<<blocks_needed, THREADS_PER_BLOCK, 0, stream>>>(
      histogram, tree.node_sums, num_features, num_outputs, split_proposals, batch);
    CHECK_CUDA_STREAM(stream);
  }

  void PerformBestSplit(Tree& tree,
                        Histogram<IntegerGPair> histogram,
                        double alpha,
                        NodeBatch batch)
  {
    perform_best_split<<<batch.NodesInBatch(), THREADS_PER_BLOCK, 0, stream>>>(histogram,
                                                                               num_features,
                                                                               num_outputs,
                                                                               split_proposals,
                                                                               eps,
                                                                               alpha,
                                                                               tree.leaf_value,
                                                                               tree.node_sums,
                                                                               tree.feature,
                                                                               tree.split_value,
                                                                               tree.gain,
                                                                               batch,
                                                                               quantiser);
    CHECK_CUDA_STREAM(stream);
  }
  void InitialiseRoot(legate::TaskContext context,
                      Tree& tree,
                      legate::AccessorRO<double, 3> g,
                      legate::AccessorRO<double, 3> h,
                      legate::Rect<3> g_shape,
                      double alpha)
  {
    const size_t blocks = (num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 grid_shape     = dim3(blocks, num_outputs);
    reduce_base_sums<<<grid_shape, THREADS_PER_BLOCK, 0, stream>>>(
      g, h, num_rows, g_shape.lo[0], tree.node_sums, num_outputs, quantiser);
    CHECK_CUDA_STREAM(stream);

    SumAllReduce(
      context, reinterpret_cast<int64_t*>(tree.node_sums.ptr({0, 0})), num_outputs * 2, stream);
    LaunchN(num_outputs,
            stream,
            [            =,
             num_outputs = this->num_outputs,
             leaf_value  = tree.leaf_value,
             node_sums   = tree.node_sums,
             quantiser   = this->quantiser] __device__(int output) {
              GPair sum               = quantiser.Dequantise(node_sums[{0, output}]);
              leaf_value[{0, output}] = CalculateLeafValue(sum.grad, sum.hess, alpha);
            });
    CHECK_CUDA_STREAM(stream);
  }

  // Create a new histogram for this batch if we need to
  // Destroy the old one
  Histogram<IntegerGPair> GetHistogram(NodeBatch batch)
  {
    if (histogram.ContainsBatch(batch.node_idx_begin, batch.node_idx_end)) { return histogram; }

    CHECK_CUDA(hipStreamSynchronize(stream));
    histogram.Destroy();
    histogram = Histogram<IntegerGPair>(batch.node_idx_begin,
                                        batch.node_idx_end,
                                        num_outputs,
                                        split_proposals.histogram_size,
                                        stream);
    return histogram;
  }

  template <typename PolicyT>
  std::vector<NodeBatch> PrepareBatches(int depth, PolicyT& policy)
  {
    // Shortcut if we have 1 batch
    if (BinaryTree::NodesInLevel(depth) <= max_batch_size) {
      // All instances are in batch
      return {NodeBatch{BinaryTree::LevelBegin(depth),
                        BinaryTree::LevelEnd(depth),
                        sorted_positions.ptr(0),
                        sorted_positions.ptr(0) + num_rows}};
    }

    thrust::sort(
      policy,
      sorted_positions.ptr(0),
      sorted_positions.ptr(num_rows),
      [] __device__(auto a, auto b) { return cuda::std::get<0>(a) < cuda::std::get<0>(b); });

    // Launch a kernel where each thread computes the range of instances for a batch using binary
    // search
    const int num_batches = (BinaryTree::NodesInLevel(depth) + max_batch_size - 1) / max_batch_size;
    auto batches          = legate::create_buffer<NodeBatch, 1>(num_batches);
    LaunchN(num_batches,
            stream,
            [                     =,
             batches_ptr          = batches.ptr(0),
             sorted_positions_ptr = this->sorted_positions.ptr(0),
             num_rows             = this->num_rows,
             max_batch_size       = this->max_batch_size] __device__(int batch_idx) {
              int batch_begin = BinaryTree::LevelBegin(depth) + batch_idx * max_batch_size;
              int batch_end   = std::min(batch_begin + max_batch_size, BinaryTree::LevelEnd(depth));
              auto comp       = [] __device__(auto a, auto b) {
                return cuda::std::get<0>(a) < cuda::std::get<0>(b);
              };

              auto lower             = thrust::lower_bound(thrust::seq,
                                               sorted_positions_ptr,
                                               sorted_positions_ptr + num_rows,
                                               cuda::std::tuple(batch_begin, 0),
                                               comp);
              auto upper             = thrust::upper_bound(thrust::seq,
                                               lower,
                                               sorted_positions_ptr + num_rows,
                                               cuda::std::tuple(batch_end - 1, 0),
                                               comp);
              batches_ptr[batch_idx] = {batch_begin, batch_end, lower, upper};
            });

    std::vector<NodeBatch> result(num_batches);
    CHECK_CUDA(hipMemcpyAsync(result.data(),
                               batches.ptr(0),
                               num_batches * sizeof(NodeBatch),
                               hipMemcpyDeviceToHost,
                               stream));
    CHECK_CUDA(hipStreamSynchronize(stream));
    // Filter empty
    result.erase(
      std::remove_if(
        result.begin(), result.end(), [](const NodeBatch& b) { return b.InstancesInBatch() == 0; }),
      result.end());
    return result;
  }

  legate::Buffer<cuda::std::tuple<int32_t, int32_t>> sorted_positions;  // (node, row)
  const int32_t num_rows;
  const int32_t num_features;
  const int32_t num_outputs;
  const int32_t max_nodes;
  SparseSplitProposals<T> split_proposals;
  Histogram<IntegerGPair> histogram;
  int max_batch_size;
  GradientQuantiser quantiser;

  hipStream_t stream;
};

struct build_tree_fn {
  template <typename T>
  void operator()(legate::TaskContext context)
  {
    auto [X, X_shape, X_accessor] = GetInputStore<T, 3>(context.input(0).data());
    auto [g, g_shape, g_accessor] = GetInputStore<double, 3>(context.input(1).data());
    auto [h, h_shape, h_accessor] = GetInputStore<double, 3>(context.input(2).data());

    EXPECT_DENSE_ROW_MAJOR(X_accessor.accessor, X_shape);
    auto num_features = X_shape.hi[1] - X_shape.lo[1] + 1;
    auto num_rows     = std::max<int64_t>(X_shape.hi[0] - X_shape.lo[0] + 1, 0);
    auto num_outputs  = X_shape.hi[2] - X_shape.lo[2] + 1;
    EXPECT(g_shape.lo[2] == 0, "Outputs should not be split between workers.");
    EXPECT_AXIS_ALIGNED(0, X_shape, g_shape);
    EXPECT_AXIS_ALIGNED(0, g_shape, h_shape);
    EXPECT_AXIS_ALIGNED(1, g_shape, h_shape);

    // Scalars
    auto max_depth     = context.scalars().at(0).value<int>();
    auto max_nodes     = context.scalars().at(1).value<int>();
    auto alpha         = context.scalars().at(2).value<double>();
    auto split_samples = context.scalars().at(3).value<int>();
    auto seed          = context.scalars().at(4).value<int>();
    auto dataset_rows  = context.scalars().at(5).value<int64_t>();

    auto stream             = legate::cuda::StreamPool::get_stream_pool().get_stream();
    auto thrust_alloc       = ThrustAllocator(legate::Memory::GPU_FB_MEM);
    auto thrust_exec_policy = DEFAULT_POLICY(thrust_alloc).on(stream);

    Tree tree(max_nodes, num_outputs, stream, thrust_exec_policy);

    SparseSplitProposals<T> split_proposals =
      SelectSplitSamples(context, X_accessor, X_shape, split_samples, seed, dataset_rows, stream);

    GradientQuantiser quantiser(context, g_accessor, h_accessor, g_shape, stream);

    // Begin building the tree
    TreeBuilder<T> builder(num_rows,
                           num_features,
                           num_outputs,
                           stream,
                           tree.max_nodes,
                           max_depth,
                           split_proposals,
                           quantiser);

    builder.InitialiseRoot(context, tree, g_accessor, h_accessor, g_shape, alpha);

    for (int depth = 0; depth < max_depth; ++depth) {
      auto batches = builder.PrepareBatches(depth, thrust_exec_policy);
      for (auto batch : batches) {
        auto histogram = builder.GetHistogram(batch);

        builder.ComputeHistogram(
          histogram, context, tree, X_accessor, X_shape, g_accessor, h_accessor, batch);

        builder.PerformBestSplit(tree, histogram, alpha, batch);
      }
      // Update position of entire level
      // Don't bother updating positions for the last level
      if (depth < max_depth - 1) { builder.UpdatePositions(tree, X_accessor, X_shape); }
    }

    tree.WriteTreeOutput(context, thrust_exec_policy, quantiser);

    CHECK_CUDA(hipStreamSynchronize(stream));
    CHECK_CUDA_STREAM(stream);
  }
};

}  // namespace

/*static*/ void BuildTreeTask::gpu_variant(legate::TaskContext context)
{
  const auto& X = context.input(0).data();
  type_dispatch_float(X.code(), build_tree_fn(), context);
}

}  // namespace legateboost
