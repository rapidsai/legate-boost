#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include "legate_library.h"
#include "legateboost.h"
#include "../../cpp_utils/cpp_utils.h"
#include "../../cpp_utils/cpp_utils.cuh"
#include "core/comm/coll.h"
#include "build_tree.h"
#include <numeric>

#include <cuda/std/tuple>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <thrust/unique.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

namespace legateboost {

__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  reduce_base_sums(legate::AccessorRO<double, 3> g,
                   legate::AccessorRO<double, 3> h,
                   size_t n_local_samples,
                   int64_t sample_offset,
                   legate::Buffer<double, 1> base_sums,
                   size_t n_outputs)
{
  typedef hipcub::BlockReduce<double, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage_g;
  __shared__ typename BlockReduce::TempStorage temp_storage_h;

  int32_t output = blockIdx.y;

  int64_t sample_id = threadIdx.x + blockDim.x * blockIdx.x;

  double G = sample_id < n_local_samples ? g[{sample_id + sample_offset, 0, output}] : 0.0;
  double H = sample_id < n_local_samples ? h[{sample_id + sample_offset, 0, output}] : 0.0;

  double blocksumG = BlockReduce(temp_storage_g).Sum(G);
  double blocksumH = BlockReduce(temp_storage_h).Sum(H);

  if (threadIdx.x == 0) {
    atomicAdd(&base_sums[output], blocksumG);
    atomicAdd(&base_sums[output + n_outputs], blocksumH);
  }
}

template <typename TYPE, int ELEMENTS_PER_THREAD, int FEATURES_PER_BLOCK>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  fill_histogram(legate::AccessorRO<TYPE, 3> X,
                 size_t n_local_samples,
                 size_t n_features,
                 int64_t sample_offset,
                 legate::AccessorRO<double, 3> g,
                 legate::AccessorRO<double, 3> h,
                 size_t n_outputs,
                 SparseSplitProposals<TYPE> split_proposals,
                 int32_t* positions_local,
                 legate::Buffer<GPair, 3> histogram,
                 legate::Buffer<double, 2> node_hessians,
                 int depth)
{
  // block dimensions are (THREADS_PER_BLOCK, 1, 1)
  // each thread processes ELEMENTS_PER_THREAD samples and FEATURES_PER_BLOCK features
  // the features to process are defined via blockIdx.y

  // further improvements:
  // * quantize values to work with int instead of double

#pragma unroll
  for (int32_t elementIdx = 0; elementIdx < ELEMENTS_PER_THREAD; ++elementIdx) {
    // within each iteration a (THREADS_PER_BLOCK, FEATURES_PER_BLOCK)-block of
    // data from X is processed.

    // check if thread has actual work to do
    int32_t localSampleId = (blockIdx.x + elementIdx * gridDim.x) * THREADS_PER_BLOCK + threadIdx.x;
    int64_t globalSampleId = localSampleId + sample_offset;
    bool validThread       = localSampleId < n_local_samples;
    if (!validThread) continue;

    int32_t sampleNode    = positions_local[localSampleId];
    bool computeHistogram = ComputeHistogramBin(sampleNode, depth, node_hessians);

    for (int32_t output = 0; output < n_outputs; output++) {
      double G = g[{globalSampleId, 0, output}];
      double H = h[{globalSampleId, 0, output}];
      for (int32_t featureIdx = 0; featureIdx < FEATURES_PER_BLOCK; featureIdx++) {
        int32_t feature = featureIdx + blockIdx.y * FEATURES_PER_BLOCK;
        if (computeHistogram && feature < n_features) {
          auto x_value = X[{globalSampleId, feature, 0}];
          auto bin_idx = split_proposals.FindBin(x_value, feature);

          // bin_idx is the first sample that is larger than x_value
          if (bin_idx != SparseSplitProposals<TYPE>::NOT_FOUND) {
            double* addPosition =
              reinterpret_cast<double*>(&histogram[{sampleNode, output, bin_idx}]);
            atomicAdd(addPosition, G);
            atomicAdd(addPosition + 1, H);
          }
        }
      }
    }
  }
}

template <typename T>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK)
  scan_kernel(legate::Buffer<GPair, 3> histogram,
              legate::Buffer<double, 2> node_hessians,
              int n_features,
              int n_outputs,
              const SparseSplitProposals<T> split_proposals,
              int depth,
              int num_nodes_to_process)

{
  auto warp = cg::tiled_partition<32>(cg::this_thread_block());
  int rank  = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int i     = rank / num_nodes_to_process;
  int j     = rank % num_nodes_to_process;

  // Specialize WarpScan for type int
  typedef hipcub::WarpScan<GPair> WarpScan;

  __shared__ typename WarpScan::TempStorage temp_storage[THREADS_PER_BLOCK / 32];

  if (i >= n_features) return;

  int scan_node_idx, subtract_node_idx;
  if (depth == 0) {
    scan_node_idx     = 0;
    subtract_node_idx = -1;
  } else {
    int parent_idx    = BinaryTree::LevelBegin(depth - 1) + j;
    auto [scan, sub]  = SelectHistogramNode(parent_idx, node_hessians);
    scan_node_idx     = scan;
    subtract_node_idx = sub;
  }

  int feature_idx                   = i;
  auto [feature_begin, feature_end] = split_proposals.FeatureRange(feature_idx);
  int num_bins                      = feature_end - feature_begin;
  int num_tiles                     = (num_bins + warp.num_threads() - 1) / warp.num_threads();

  for (int output = 0; output < n_outputs; output++) {
    GPair aggregate;
    // Scan left side
    for (int tile_idx = 0; tile_idx < num_tiles; tile_idx++) {
      int bin_idx              = feature_begin + tile_idx * warp.num_threads() + warp.thread_rank();
      bool thread_participates = bin_idx < feature_end;
      auto e = thread_participates ? histogram[{scan_node_idx, output, bin_idx}] : GPair{0, 0};
      GPair tile_aggregate;
      WarpScan(temp_storage[threadIdx.x / warp.num_threads()]).InclusiveSum(e, e, tile_aggregate);
      __syncwarp();
      if (thread_participates) { histogram[{scan_node_idx, output, bin_idx}] = e + aggregate; }
      aggregate += tile_aggregate;
    }
  }

  if (depth == 0) return;

  for (int output = 0; output < n_outputs; output++) {
    // Infer right side
    for (int bin_idx = feature_begin + warp.thread_rank(); bin_idx < feature_end;
         bin_idx += warp.num_threads()) {
      GPair scanned_sum = histogram[{scan_node_idx, output, bin_idx}];
      GPair parent_sum  = histogram[{BinaryTree::Parent(scan_node_idx), output, bin_idx}];
      GPair other_sum   = parent_sum - scanned_sum;
      histogram[{subtract_node_idx, output, bin_idx}] = other_sum;
    }
  }
}
// Key/value pair to simplify reduction
struct GainFeaturePair {
  double gain;
  int feature;
  int feature_sample_idx;

  __device__ void operator=(const GainFeaturePair& other)
  {
    gain               = other.gain;
    feature            = other.feature;
    feature_sample_idx = other.feature_sample_idx;
  }

  __device__ bool operator==(const GainFeaturePair& other) const
  {
    return gain == other.gain && feature == other.feature &&
           feature_sample_idx == other.feature_sample_idx;
  }

  __device__ bool operator>(const GainFeaturePair& other) const { return gain > other.gain; }

  __device__ bool operator<(const GainFeaturePair& other) const { return gain < other.gain; }
};

template <typename TYPE>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  perform_best_split(legate::Buffer<GPair, 3> histogram,
                     size_t n_features,
                     size_t n_outputs,
                     SparseSplitProposals<TYPE> split_proposals,
                     double eps,
                     double alpha,
                     legate::Buffer<double, 2> tree_leaf_value,
                     legate::Buffer<double, 2> tree_gradient,
                     legate::Buffer<double, 2> tree_hessian,
                     legate::Buffer<int32_t, 1> tree_feature,
                     legate::Buffer<double, 1> tree_split_value,
                     legate::Buffer<double, 1> tree_gain,
                     int depth)
{
  // using one block per (level) node to have blockwise reductions
  int node_id = blockIdx.x + BinaryTree::LevelBegin(depth);

  typedef hipcub::BlockReduce<GainFeaturePair, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  __shared__ double node_best_gain;
  __shared__ int node_best_feature;
  __shared__ int node_best_bin_idx;

  double thread_best_gain = 0;
  int thread_best_feature = -1;
  int thread_best_bin_idx = -1;

  for (int feature_id = 0; feature_id < n_features; feature_id++) {
    auto [feature_start, feature_end] = split_proposals.FeatureRange(feature_id);

    for (int bin_idx = feature_start + threadIdx.x; bin_idx < feature_end; bin_idx += blockDim.x) {
      double gain = 0;
      for (int output = 0; output < n_outputs; ++output) {
        auto G          = tree_gradient[{node_id, output}];
        auto H          = tree_hessian[{node_id, output}];
        auto [G_L, H_L] = histogram[{node_id, output, bin_idx}];
        auto G_R        = G - G_L;
        auto H_R        = H - H_L;

        if (H_L <= 0.0 || H_R <= 0.0) {
          gain = 0;
          break;
        }
        double reg = std::max(eps, alpha);  // Regularisation term
        gain += 0.5 * ((G_L * G_L) / (H_L + reg) + (G_R * G_R) / (H_R + reg) - (G * G) / (H + reg));
      }
      if (gain > thread_best_gain) {
        thread_best_gain    = gain;
        thread_best_feature = feature_id;
        thread_best_bin_idx = bin_idx;
      }
    }
  }

  // SYNC BEST GAIN TO FULL BLOCK/NODE
  GainFeaturePair thread_best_pair{thread_best_gain, thread_best_feature, thread_best_bin_idx};
  GainFeaturePair node_best_pair =
    BlockReduce(temp_storage).Reduce(thread_best_pair, hipcub::Max(), THREADS_PER_BLOCK);
  if (threadIdx.x == 0) {
    node_best_gain    = node_best_pair.gain;
    node_best_feature = node_best_pair.feature;
    node_best_bin_idx = node_best_pair.feature_sample_idx;
  }
  __syncthreads();

  if (node_best_gain > eps) {
    for (int output = threadIdx.x; output < n_outputs; output += blockDim.x) {
      auto [G_L, H_L] = histogram[{node_id, output, node_best_bin_idx}];
      auto G_R        = tree_gradient[{node_id, output}] - G_L;
      auto H_R        = tree_hessian[{node_id, output}] - H_L;

      int left_child                         = BinaryTree::LeftChild(node_id);
      int right_child                        = BinaryTree::RightChild(node_id);
      tree_leaf_value[{left_child, output}]  = CalculateLeafValue(G_L, H_L, alpha);
      tree_leaf_value[{right_child, output}] = CalculateLeafValue(G_R, H_R, alpha);
      tree_hessian[{left_child, output}]     = H_L;
      tree_hessian[{right_child, output}]    = H_R;
      tree_gradient[{left_child, output}]    = G_L;
      tree_gradient[{right_child, output}]   = G_R;

      if (output == 0) {
        tree_feature[node_id]     = node_best_feature;
        tree_split_value[node_id] = split_proposals.split_proposals[{node_best_bin_idx}];
        tree_gain[node_id]        = node_best_gain;
      }
    }
  }
}

namespace {

struct Tree {
  template <typename THRUST_POLICY>
  Tree(int max_nodes, int num_outputs, hipStream_t stream, const THRUST_POLICY& thrust_exec_policy)
    : num_outputs(num_outputs), max_nodes(max_nodes), stream(stream)
  {
    leaf_value  = legate::create_buffer<double, 2>({max_nodes, num_outputs});
    feature     = legate::create_buffer<int32_t, 1>({max_nodes});
    split_value = legate::create_buffer<double, 1>({max_nodes});
    gain        = legate::create_buffer<double, 1>({max_nodes});
    hessian     = legate::create_buffer<double, 2>({max_nodes, num_outputs});
    gradient    = legate::create_buffer<double, 2>({max_nodes, num_outputs});
    thrust::fill(thrust_exec_policy,
                 leaf_value.ptr({0, 0}),
                 leaf_value.ptr({0, 0}) + max_nodes * num_outputs,
                 0.0);
    thrust::fill(thrust_exec_policy, feature.ptr({0}), feature.ptr({0}) + max_nodes, -1);
    thrust::fill(
      thrust_exec_policy, hessian.ptr({0, 0}), hessian.ptr({0, 0}) + max_nodes * num_outputs, 0.0);
    thrust::fill(thrust_exec_policy, split_value.ptr({0}), split_value.ptr({0}) + max_nodes, 0.0);
    thrust::fill(thrust_exec_policy, gain.ptr({0}), gain.ptr({0}) + max_nodes, 0.0);
    thrust::fill(thrust_exec_policy,
                 gradient.ptr({0, 0}),
                 gradient.ptr({0, 0}) + max_nodes * num_outputs,
                 0.0);
  }

  ~Tree()
  {
    leaf_value.destroy();
    feature.destroy();
    split_value.destroy();
    gain.destroy();
    hessian.destroy();
    gradient.destroy();
  }

  void InitializeBase(legate::Buffer<double, 1> base_sums, double alpha)
  {
    LaunchN(num_outputs,
            stream,
            [            =,
             num_outputs = this->num_outputs,
             leaf_value  = this->leaf_value,
             gradient    = this->gradient,
             hessian     = this->hessian] __device__(int output) {
              leaf_value[{0, output}] =
                CalculateLeafValue(base_sums[output], base_sums[output + num_outputs], alpha);
              gradient[{0, output}] = base_sums[output];
              hessian[{0, output}]  = base_sums[output + num_outputs];
            });
  }

  template <typename T, int DIM, typename ThrustPolicyT>
  void WriteOutput(legate::PhysicalStore out,
                   const legate::Buffer<T, DIM> x,
                   const ThrustPolicyT& policy)
  {
    // Write a tile of x to the output
    const legate::Rect<DIM> out_shape = out.shape<DIM>();
    auto out_acc                      = out.write_accessor<T, DIM>();
    thrust::for_each_n(policy,
                       UnravelIter(out_shape),
                       out_shape.volume(),
                       [=] __host__ __device__(const legate::Point<DIM>& p) { out_acc[p] = x[p]; });
  }

  template <typename ThrustPolicyT>
  void WriteTreeOutput(legate::TaskContext context, const ThrustPolicyT& policy)
  {
    WriteOutput(context.output(0).data(), leaf_value, policy);
    WriteOutput(context.output(1).data(), feature, policy);
    WriteOutput(context.output(2).data(), split_value, policy);
    WriteOutput(context.output(3).data(), gain, policy);
    WriteOutput(context.output(4).data(), hessian, policy);
    CHECK_CUDA_STREAM(stream);
  }

  legate::Buffer<double, 2> leaf_value;
  legate::Buffer<int32_t, 1> feature;
  legate::Buffer<double, 1> split_value;
  legate::Buffer<double, 1> gain;
  legate::Buffer<double, 2> hessian;
  legate::Buffer<double, 2> gradient;
  const int num_outputs;
  const int max_nodes;
  hipStream_t stream;
};

// Randomly sample split_samples rows from X
// Use nccl to share the samples with all workers
// Remove any duplicates
// Return sparse matrix of split samples for each feature
template <typename T>
SparseSplitProposals<T> SelectSplitSamples(legate::TaskContext context,
                                           legate::AccessorRO<T, 3> X,
                                           legate::Rect<3> X_shape,
                                           int split_samples,
                                           int seed,
                                           int64_t dataset_rows,
                                           hipStream_t stream)
{
  auto thrust_alloc = ThrustAllocator(legate::Memory::GPU_FB_MEM);
  auto policy       = DEFAULT_POLICY(thrust_alloc).on(stream);
  int num_features  = X_shape.hi[1] - X_shape.lo[1] + 1;
  // Randomly choose split_samples rows
  auto row_samples = legate::create_buffer<int64_t, 1>(split_samples);
  auto counting    = thrust::make_counting_iterator(0);
  thrust::transform(
    policy, counting, counting + split_samples, row_samples.ptr(0), [=] __device__(int64_t idx) {
      thrust::default_random_engine eng(seed);
      thrust::uniform_int_distribution<int64_t> dist(0, dataset_rows - 1);
      eng.discard(idx);
      return dist(eng);
    });
  auto draft_proposals = legate::create_buffer<T, 2>({num_features, split_samples});

  // fill with local data
  LaunchN(num_features * split_samples, stream, [=] __device__(auto idx) {
    auto i                  = idx / num_features;
    auto j                  = idx % num_features;
    auto row                = row_samples[i];
    bool has_data           = row >= X_shape.lo[0] && row <= X_shape.hi[0];
    draft_proposals[{j, i}] = has_data ? X[{row, j, 0}] : T(0);
  });

  // Sum reduce over all workers
  SumAllReduce(context, draft_proposals.ptr({0, 0}), num_features * split_samples, stream);

  CHECK_CUDA_STREAM(stream);

  // Condense split samples to unique values
  // First sort the samples
  auto keys = legate::create_buffer<int32_t, 1>(num_features * split_samples);
  thrust::transform(
    policy, counting, counting + num_features * split_samples, keys.ptr(0), [=] __device__(int i) {
      return i / split_samples;
    });

  // Segmented sort
  auto begin =
    thrust::make_zip_iterator(thrust::make_tuple(keys.ptr(0), draft_proposals.ptr({0, 0})));
  thrust::sort(policy, begin, begin + num_features * split_samples, [] __device__(auto a, auto b) {
    if (thrust::get<0>(a) != thrust::get<0>(b)) { return thrust::get<0>(a) < thrust::get<0>(b); }
    return thrust::get<1>(a) < thrust::get<1>(b);
  });

  // Extract the unique values
  auto out_keys        = legate::create_buffer<int32_t, 1>(num_features * split_samples);
  auto split_proposals = legate::create_buffer<T, 1>(num_features * split_samples);
  auto key_val =
    thrust::make_zip_iterator(thrust::make_tuple(keys.ptr(0), draft_proposals.ptr({0, 0})));
  auto out_iter =
    thrust::make_zip_iterator(thrust::make_tuple(out_keys.ptr(0), split_proposals.ptr(0)));
  auto result =
    thrust::unique_copy(policy, key_val, key_val + num_features * split_samples, out_iter);
  auto n_unique = thrust::distance(out_iter, result);
  // Count the unique values for each feature
  auto row_pointers = legate::create_buffer<int32_t, 1>(num_features + 1);
  CHECK_CUDA(hipMemsetAsync(row_pointers.ptr(0), 0, (num_features + 1) * sizeof(int32_t), stream));

  thrust::reduce_by_key(policy,
                        out_keys.ptr(0),
                        out_keys.ptr(0) + n_unique,
                        thrust::make_constant_iterator(1),
                        thrust::make_discard_iterator(),
                        row_pointers.ptr(1));
  // Scan the counts to get the row pointers for a CSR matrix
  thrust::inclusive_scan(
    policy, row_pointers.ptr(1), row_pointers.ptr(1) + num_features, row_pointers.ptr(1));

  CHECK_CUDA(hipStreamSynchronize(stream));
  row_samples.destroy();
  draft_proposals.destroy();
  out_keys.destroy();
  return SparseSplitProposals<T>(split_proposals, row_pointers, num_features, n_unique);
}
template <typename T>
struct TreeBuilder {
  TreeBuilder(int32_t num_rows,
              int32_t num_features,
              int32_t num_outputs,
              hipStream_t stream,
              int32_t max_nodes,
              SparseSplitProposals<T> split_proposals)
    : num_rows(num_rows),
      num_features(num_features),
      num_outputs(num_outputs),
      stream(stream),
      max_nodes(max_nodes),
      split_proposals(split_proposals)
  {
    positions = legate::create_buffer<int32_t>(num_rows);
    histogram_buffer =
      legate::create_buffer<GPair, 3>({max_nodes, num_outputs, split_proposals.histogram_size});
    CHECK_CUDA(
      hipMemsetAsync(histogram_buffer.ptr(legate::Point<3>::ZEROES()),
                      0,
                      max_nodes * num_outputs * split_proposals.histogram_size * sizeof(GPair),
                      stream));
    // some initialization on first pass
    CHECK_CUDA(hipMemsetAsync(positions.ptr(0), 0, (size_t)num_rows * sizeof(int32_t), stream));
  }

  ~TreeBuilder()
  {
    positions.destroy();
    histogram_buffer.destroy();
    if (cub_buffer_size > 0) cub_buffer.destroy();
  }

  template <typename TYPE>
  void UpdatePositions(int depth,
                       Tree& tree,
                       legate::AccessorRO<TYPE, 3> X,
                       legate::Rect<3> X_shape)
  {
    if (depth == 0) return;
    auto tree_split_value_ptr    = tree.split_value.ptr(0);
    auto tree_feature_ptr        = tree.feature.ptr(0);
    auto positions_ptr           = positions.ptr(0);
    auto max_nodes_              = this->max_nodes;
    auto update_positions_lambda = [=] __device__(size_t idx) {
      int32_t& pos = positions_ptr[idx];
      if (pos < 0 || pos >= max_nodes_ || tree_feature_ptr[pos] == -1) {
        pos = -1;
        return;
      }
      double x_value = X[{X_shape.lo[0] + (int64_t)idx, tree_feature_ptr[pos], 0}];
      bool left      = x_value <= tree_split_value_ptr[pos];
      pos            = left ? BinaryTree::LeftChild(pos) : BinaryTree::RightChild(pos);
    };
    LaunchN(num_rows, stream, update_positions_lambda);
    CHECK_CUDA_STREAM(stream);
  }

  template <typename TYPE>
  void ComputeHistogram(int depth,
                        legate::TaskContext context,
                        Tree& tree,
                        legate::AccessorRO<TYPE, 3> X,
                        legate::Rect<3> X_shape,
                        legate::AccessorRO<double, 3> g,
                        legate::AccessorRO<double, 3> h)
  {
    // TODO adjust kernel parameters dynamically
    constexpr size_t elements_per_thread = 8;
    constexpr size_t features_per_block  = 16;
    const size_t blocks_x = (num_rows + THREADS_PER_BLOCK * elements_per_thread - 1) /
                            (THREADS_PER_BLOCK * elements_per_thread);
    const size_t blocks_y = (num_features + features_per_block - 1) / features_per_block;
    dim3 grid_shape       = dim3(blocks_x, blocks_y, 1);
    fill_histogram<TYPE, elements_per_thread, features_per_block>
      <<<grid_shape, THREADS_PER_BLOCK, 0, stream>>>(X,
                                                     num_rows,
                                                     num_features,
                                                     X_shape.lo[0],
                                                     g,
                                                     h,
                                                     num_outputs,
                                                     split_proposals,
                                                     positions.ptr(0),
                                                     histogram_buffer,
                                                     tree.hessian,
                                                     depth);
    CHECK_CUDA_STREAM(stream);
    static_assert(sizeof(GPair) == 2 * sizeof(double), "GPair must be 2 doubles");
    SumAllReduce(
      context,
      reinterpret_cast<double*>(histogram_buffer.ptr({BinaryTree::LevelBegin(depth), 0, 0})),
      BinaryTree::NodesInLevel(depth) * num_outputs * split_proposals.histogram_size * 2,
      stream);

    const int num_nodes_to_process = std::max(BinaryTree::NodesInLevel(depth) / 2, 1);
    const size_t warps_needed      = num_features * num_nodes_to_process;
    const size_t warps_per_block   = THREADS_PER_BLOCK / 32;
    const size_t blocks_needed     = (warps_needed + warps_per_block - 1) / warps_per_block;

    // Scan the histogram
    // Then do subtraction trick to infer right side from parent and left side
    scan_kernel<<<blocks_needed, THREADS_PER_BLOCK, 0, stream>>>(histogram_buffer,
                                                                 tree.hessian,
                                                                 num_features,
                                                                 num_outputs,
                                                                 split_proposals,
                                                                 depth,
                                                                 num_nodes_to_process);
    CHECK_CUDA_STREAM(stream);
  }

  void PerformBestSplit(int depth, Tree& tree, double alpha)
  {
    perform_best_split<<<BinaryTree::NodesInLevel(depth), THREADS_PER_BLOCK, 0, stream>>>(
      histogram_buffer,
      num_features,
      num_outputs,
      split_proposals,
      eps,
      alpha,
      tree.leaf_value,
      tree.gradient,
      tree.hessian,
      tree.feature,
      tree.split_value,
      tree.gain,
      depth);
    CHECK_CUDA_STREAM(stream);
  }
  void InitialiseRoot(legate::TaskContext context,
                      Tree& tree,
                      legate::AccessorRO<double, 3> g,
                      legate::AccessorRO<double, 3> h,
                      legate::Rect<3> g_shape,
                      double alpha)
  {
    auto base_sums = legate::create_buffer<double, 1>(num_outputs * 2);

    CHECK_CUDA(hipMemsetAsync(base_sums.ptr(0), 0, num_outputs * 2 * sizeof(double), stream));
    const size_t blocks = (num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 grid_shape     = dim3(blocks, num_outputs);
    reduce_base_sums<<<grid_shape, THREADS_PER_BLOCK, 0, stream>>>(
      g, h, num_rows, g_shape.lo[0], base_sums, num_outputs);
    CHECK_CUDA_STREAM(stream);

    SumAllReduce(context, reinterpret_cast<double*>(base_sums.ptr(0)), num_outputs * 2, stream);

    // base sums contain g-sums first, h sums second
    tree.InitializeBase(base_sums, alpha);

    base_sums.destroy();
    CHECK_CUDA_STREAM(stream);
  }

  legate::Buffer<int32_t> positions;
  const int32_t num_rows;
  const int32_t num_features;
  const int32_t num_outputs;
  const int32_t max_nodes;
  SparseSplitProposals<T> split_proposals;

  legate::Buffer<unsigned char> cub_buffer;
  size_t cub_buffer_size = 0;

  legate::Buffer<GPair, 3> histogram_buffer;

  hipStream_t stream;
};

struct build_tree_fn {
  template <typename T>
  void operator()(legate::TaskContext context)
  {
    auto [X, X_shape, X_accessor] = GetInputStore<T, 3>(context.input(0).data());
    auto [g, g_shape, g_accessor] = GetInputStore<double, 3>(context.input(1).data());
    auto [h, h_shape, h_accessor] = GetInputStore<double, 3>(context.input(2).data());

    EXPECT_DENSE_ROW_MAJOR(X_accessor.accessor, X_shape);
    auto num_features = X_shape.hi[1] - X_shape.lo[1] + 1;
    auto num_rows     = std::max<int64_t>(X_shape.hi[0] - X_shape.lo[0] + 1, 0);
    auto num_outputs  = X_shape.hi[2] - X_shape.lo[2] + 1;
    EXPECT(g_shape.lo[2] == 0, "Outputs should not be split between workers.");
    EXPECT_AXIS_ALIGNED(0, X_shape, g_shape);
    EXPECT_AXIS_ALIGNED(0, g_shape, h_shape);
    EXPECT_AXIS_ALIGNED(1, g_shape, h_shape);

    // Scalars
    auto max_depth     = context.scalars().at(0).value<int>();
    auto max_nodes     = context.scalars().at(1).value<int>();
    auto alpha         = context.scalars().at(2).value<double>();
    auto split_samples = context.scalars().at(3).value<int>();
    auto seed          = context.scalars().at(4).value<int>();
    auto dataset_rows  = context.scalars().at(5).value<int64_t>();

    auto stream             = legate::cuda::StreamPool::get_stream_pool().get_stream();
    auto thrust_alloc       = ThrustAllocator(legate::Memory::GPU_FB_MEM);
    auto thrust_exec_policy = DEFAULT_POLICY(thrust_alloc).on(stream);

    Tree tree(max_nodes, num_outputs, stream, thrust_exec_policy);

    SparseSplitProposals<T> split_proposals =
      SelectSplitSamples(context, X_accessor, X_shape, split_samples, seed, dataset_rows, stream);
    // Begin building the tree
    TreeBuilder<T> builder(
      num_rows, num_features, num_outputs, stream, tree.max_nodes, split_proposals);

    builder.InitialiseRoot(context, tree, g_accessor, h_accessor, g_shape, alpha);

    for (int depth = 0; depth < max_depth; ++depth) {
      // update positions from previous step
      builder.UpdatePositions(depth, tree, X_accessor, X_shape);

      // actual histogram creation
      builder.ComputeHistogram(depth, context, tree, X_accessor, X_shape, g_accessor, h_accessor);

      // Select the best split
      builder.PerformBestSplit(depth, tree, alpha);
    }

    tree.WriteTreeOutput(context, thrust_exec_policy);

    CHECK_CUDA(hipStreamSynchronize(stream));
    CHECK_CUDA_STREAM(stream);
  }
};

}  // namespace

/*static*/ void BuildTreeTask::gpu_variant(legate::TaskContext context)
{
  const auto& X = context.input(0).data();
  type_dispatch_float(X.code(), build_tree_fn(), context);
}

}  // namespace legateboost
